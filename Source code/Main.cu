#include "hip/hip_runtime.h"
// This version works with comsol files
#pragma once
#include <map>
#include <vector>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string> 
#include <sstream> 

#define _USE_MATH_DEFINES // Math constants
#include <math.h>
#include "variables.h"
#include "bending torque.h"
#include "check fiber breakage.h"
#include "SEgments parameters computation.h"
#include "Fibers Motion Solver.h"
#include "utilities.h"
#include "Find Neighbors.cuh"
#include "Excluded Volume.h"
#include "InputData.h"
#include "time integration.h"
#include "velocityMeshUtilities.h"
#define PRINT_DEBUG_INFO_GLOBAL_TIMING
//#define PRINT_DEBUG_INFO
#define IS_WINDOWS 0
#define IS_LINUX 1
#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
const int sytemOS = IS_WINDOWS;
#else
const int sytemOS = IS_LINUX;
#endif


// This is a direct translation of the mechanistic Model from Fortran to C++
// Started june/2013
using namespace std;
using namespace Eigen;


	
void initializeNamelist(){
	s_mapVariablesValues["r_bead" ]=fiber_radius_name;
	s_mapVariablesValues["viscosity" ]=viscosity_name;
	s_mapVariablesValues["dt" ]=dt_name;
	s_mapVariablesValues["nbr_intgr" ]=number_integrations_name;
	s_mapVariablesValues["writ_freq" ]=writing_frequency_name;
	s_mapVariablesValues["E_Young" ]=E_young_name;
	s_mapVariablesValues["L_seg" ]=L_seg_name;
	s_mapVariablesValues["max_alpha" ]=critical_angle_name;
	s_mapVariablesValues["break_freq" ]=check_breakage_frequency_name;
	s_mapVariablesValues["gamma_dot" ]=gamma_dot_name;
}
int readData(){

	// Approach to read namelists as fortran does but way more primitive

	// need a list of values that represent the variables names
	//string::size_type sz;     // alias of size_t
	string line;
	ifstream tempFile ("Input/Fibers.in"); // for folders use "/" do not use"\"
	
	  if (tempFile.is_open())
	  {
		  cout<<" Reading input parameters \n";
		  initializeNamelist();
		  while(getline (tempFile,line))  //read line 
		  {
			  size_t equalPosition = line.find('=');
			  if (equalPosition< line.length()) // gets rid of lines that do not have equals
			  {
				  line.replace(equalPosition, 1 , " "); //delete equal so no more steps are needed after tokenization
			  stringstream lineTokens(line); //Tokenization of line
			  string variableName;
			  lineTokens>>variableName; // first token is the variable name
			  cout<<" "<< variableName << " ";
			  switch (s_mapVariablesValues[variableName]) // look for the variable name "value" in our map and switch it
			  {
				  case gamma_dot_name:
					  lineTokens >>gamma_dot ; 
					cout<< " gamma_dot is : " << gamma_dot<< "  ";
				  break;

				   case check_breakage_frequency_name:
					  lineTokens >>check_breakage_frequency ; 
					cout<< " check_breakage_frequency is : " << check_breakage_frequency<< "  ";
				  break;

					case critical_angle_name:
					  lineTokens >>critical_angle ; 
					cout<< " critical_angle is : " << critical_angle<< "  ";
				  break;
				  case L_seg_name:
					  lineTokens >>L_segment ; 
					cout<< " L_seg is : " << L_segment<< "  ";
				  break;

				  case writing_frequency_name:
					lineTokens >>writing_frequency ; 
					cout<< " writing_frequency is : " << writing_frequency<< "  ";
				  break;

				   case number_integrations_name:
					lineTokens >>number_integrations ; 
					cout<< " number_integrations is : " << number_integrations << "  ";
				  break;

				  case dt_name:
					lineTokens >>dt ; 
					cout<< " dt is : " << dt << "  ";
				  break;

				  case viscosity_name:
					lineTokens >>viscosity ; 
					cout<< " viscosity is : " << viscosity << "  ";
				  break;

				case fiber_radius_name:
					lineTokens >>fiber_radius ; 
					cout<< " fiber_radius is : " << fiber_radius << "  ";
				  break;
			  
			  case E_young_name:
					lineTokens >>E_young ; 
					cout<< " Eyoung is : " << E_young << "  ";
				  break;
			  
			  default:
				  break;
			  }
			
			  cout<<endl;

			  }
			
		  }

		  tempFile.close();
	 }

	return 0;
}


int main(){
	

	// Initialization of the program, reading input files
	
	vector<fiber> fibers;
	
	cout<< " Mechanistic model of fibers moving in a fluid \n \n" ; 
	
	// Variables definition
	ofstream outputFile;
	ofstream framesFile;
	thrust::host_vector<int> fiberIndices;
	thrust::host_vector<int> segmentIndices;
	thrust::host_vector<long long> possibleCollisions;

	// Variables needed for 3D transient mesh flow field
	vector<double> velMeshPoints;    //Stores coordinates of all vertices in the mesh as follows
									 //X0,X1,...Xn, Y0,Y1,...,Yn, Z0,Z1,..Zn

	vector<int> velMeshConnectivity; // Stores the vertices that make up the tetrahedra
									 // each component is an node index, there are 4 index per tet
									 // N00,N01,N02,N03,N10,N11,N12,N13,...,Nn0,Nn1,n2,Nn3
	
	vector<int> velMeshNeighbors;	 // Stores the face-neighbors for each tetrahedron
									 // each component is a tet index, there are 4 index per tet
									 // N00,N01,N02,N03,N10,N11,N12,N13,...,Nn0,Nn1,n2,Nn3
									 // 3 for Tri , 6 for hex and so on
	
	vector<int> tetraHingeID;		// Stores the tet index where a certain hinge is
										
	vector<double> velTime1;		// Stores the velocity of all the nodes at a certaint time
									//Vx0,Vx1,...Vxn, Vy0,Vy1,...,Vyn, Vz0,Vz1,..Vzn
	
	vector<double> velTime2;		//same as velTime1

	vector<double> meshVorticity;		// value of vorticity at different nodes

	vector<int> meshCorners;        // in 2D nodesID of nodes that are corners, We will add statci fibers along those points

	vector<int> boundaryEdgeIndex;   // for each element stores -1 if there are no boundary edges and if there is one stores which face is boundary



	// Read Data
	outputFile.open("Output/data2.in",ios::out);
	
	readBorder(border_coords, "Input/concise_marco.txt"); // wall information is stored in border coords. TODO should be able to receive 2D or 3D
	readFlowFieldData(flowField, "Input/Coords.txt");     // Velocity field info, stored in flow field. TODO 
	readData();											  // information about the fibers 
	
	readInitialFibersPositions(fibers, hinges);// self explanatory 
	
	int nodesPerElement;

	readMeshFileTri("Input/rib_mesh.mphtxt",velMeshConnectivity,velMeshPoints,meshCorners,nodesPerElement);
	readVelFile2D("Input/fields_in_rib2.txt", velTime1,meshVorticity);
	int numNodes = velTime1.size()/3;
	int numElements = velMeshConnectivity.size()/nodesPerElement; // 6 for triangles

	generateCornerFibers(meshCorners,fibers,velMeshPoints,0.1,numNodes);
	cout<<endl<<endl<<" Simulation summary:  "<<endl;
	cout<<" Number of fibers : " << fibers.size() << " \n";
	cout<<" Number of elements : " << numElements << " \n";
	cout<<" Number of numNodes : " << numNodes << " \n";
	cout<<" Number of nodes per element : " << nodesPerElement << " \n";
	// Process the mesh
	
	velMeshNeighbors.resize(numElements*3);//3 for triangles TODO get this value from mesh Type
	

	

	boundaryEdgeIndex.resize(numElements);//assuming there is just 1 boundary per element
	find_tri_neighbors(numElements,nodesPerElement,velMeshConnectivity,velMeshNeighbors); // find neighboring elements for each element
	int elementTotest = 19;
	cout<< " the neighbors of element " <<elementTotest<< " are : "<<velMeshNeighbors[elementTotest*3+0 ]<< " "<<velMeshNeighbors[elementTotest*3+1 ]<< " "<<velMeshNeighbors[elementTotest*3+2 ]<<endl;
	
	findBoundaryEdges(numElements,3,velMeshNeighbors,boundaryEdgeIndex); //finds boundary edges indices for each element, helpful for excluded volume forces

	find_Initial_tetID(velMeshPoints,velMeshConnectivity,fibers,numElements,numNodes,2,nodesPerElement);//meshtype 2, tri elements
	

	/* initialize fiber and segment indices;
	*/
	// We need to regenerate the fiber indices and segment indices.
		int count=0;
	
		for (uint i = 0; i < fibers.size(); i++)
		{
			segmentIndices.push_back(count);
			for (int j = 0; j < fibers[i].numberOfSegments; j++)
			{
				fiberIndices.push_back(i);
				count++;
			}
		
		}


	double inertia_moment = (pi/4)*pow(fiber_radius,4);
	double * boundingBoxes;
	boundingBoxes = (double*) malloc(sizeof(double)* hinges.size() *6 );

	//main loop goes here, right now it is just one integration
	int timeA =getMilliCount();
	int timeB;

	int frames = 0;

	#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
	number_integrations=5;
	#endif
	#ifdef PRINT_DEBUG_INFO
	number_integrations=1;
	#endif

	//number_integrations =1; // Delete this line when not debugging
	for (int step = 0; step < number_integrations; step++)
	{
		if (step% check_breakage_frequency == 0 )
		{

			bendingTorqueForAllFibers(fibers,hinges,E_young, inertia_moment); // TO DO, Hinges can be removed this is bending_torque_whole
			checkFibersBreakage(fibers, critical_angle, fiberIndices, segmentIndices);         //this is  fiber_damage
			#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
			timeA =getMilliCount();
				#endif

			find_Initial_tetID(velMeshPoints,velMeshConnectivity,fibers,numElements,numNodes,2,nodesPerElement);//meshtype 2, tri elements
			#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "find_Initial_tetID done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
				
				#endif
			if (fibers.size()!= 1)
			{
				#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
				#endif

				findNeighbours(fibers,fiber_radius, possibleCollisions, fiberIndices); // To do 
								
				if (possibleCollisions.size() !=0)
				{
					refineNeighbors(fibers, fiber_radius, possibleCollisions, fiberIndices, segmentIndices);
				}
				
				#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "Find Neighbors done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
				cout<<" possible collision size" << possibleCollisions.size()<<endl;
				#endif

			}
			
		}

		

		// 
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif
				 checkHingesElements(velMeshPoints,velMeshConnectivity,velMeshNeighbors,fibers,numElements,numNodes,2,nodesPerElement);//2 for tri

		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "checkHingesElements done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif


///////////////**************************//////////////			
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif

				 //fibers_parameters_computation(fibers,fiber_radius); // fiber_par_calc
				 fibers_parameters_computation_mesh(fibers,fiber_radius,velMeshPoints,velMeshConnectivity,velTime1,meshVorticity,numElements,numNodes,nodesPerElement); // 6 is quadratic triangles
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "fibers_parameters_computation done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif

///////////////**************************//////////////			
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif

		//calculateFiberInteractions(fibers,fiber_radius,possibleCollisions,fiberIndices,segmentIndices,step); // prints debugging info
		calculateFiberInteractions(fibers,fiber_radius,possibleCollisions,fiberIndices,segmentIndices);

		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "calculateFiberInteractions done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif

#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
#endif
				 interactionsWallsTri(fibers,numNodes,velMeshPoints,velMeshConnectivity,velMeshNeighbors,fiber_radius,100,10*fiber_radius,nodesPerElement);
   #ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "interactionsWallsTri done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif

///////////////**************************//////////////	
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif

		bendingTorqueForAllFibers(fibers, hinges, E_young, inertia_moment);
		
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "bendingTorqueForAllFibers done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif

///////////////**************************//////////////	
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif

			solveFiberMotion(fibers,fiber_radius,step); // mot

		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "solveFiberMotion done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif
		
///////////////**************************//////////////	
		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				 timeA =getMilliCount();
		#endif

		updatePositions(fibers,dt);
		

		#ifdef PRINT_DEBUG_INFO_GLOBAL_TIMING
				cout << "updatePositions done, time elpased:  " << getMilliSpan(timeA)<<" ms"<<endl;
	    #endif
///////////////**************************//////////////	

		#ifdef PRINT_DEBUG_INFO
		for (int j= 0; j < fibers[0].numberOfHinges; j++)
		{
			cout<<endl<<endl;
			cout<< " Hinge " << j<< " r_times_u_sum "<< fibers[0].hinges[j].r_times_u_sum<<endl;
			cout<< " Hinge " << j<< " fluid_velocity_sum "<< fibers[0].hinges[j].fluid_velocity_sum<<endl;
			cout<< " Hinge " << j<< " sumOmegaFluid "<< fibers[0].hinges[j].sumOmegaFluid<<endl;
			cout<< " Hinge " << j<< " averageviscosity "<< fibers[0].hinges[j].averageviscosity<<endl;
			cout<< " Hinge " << j<< " r_sum "<< fibers[0].hinges[j].r_sum<<endl;
			cout<< " Hinge " << j<< " r_product_sum "<< fibers[0].hinges[j].r_product_sum<<endl;
			cout<< " Hinge " << j<< " exluded_volume_force "<< fibers[0].hinges[j].exluded_volume_force<<endl;
			cout<< " Hinge " << j<< " excluded_volume_torque "<< fibers[0].hinges[j].excluded_volume_torque<<endl;
			cout<< " Hinge " << j<< " torque "<< fibers[0].hinges[j].torque<<endl;
			cout<< " Hinge " << j<< " velocity "<< fibers[0].hinges[j].velocity<<endl;
			cout<< " Hinge " << j<< " position "<< fibers[0].hinges[j].position<<endl;
			int elementID = fibers[0].hinges[j].elementID;
			cout<< " Hinge " << j << " Element " << elementID<<endl;
			cout<< " nodes "<< " : " ;
				for (int kk = 0; kk < nodesPerElement; kk++)
				{
					cout << velMeshConnectivity[elementID*nodesPerElement+kk]<<" ";
				}
				cout<<endl;
			



		}
		
		cout<<endl;
		#endif
		
		//cout<<" Time needed for updatePositions is : " << getMilliSpan(timeB) << "ms" << endl;
		//cout<<" Integration number:  " << step<<endl; 
		// Write output file
		if (step%writing_frequency == 0)
		{
			
			frames ++;
			printPositions(fibers,outputFile,frames);
			cout<<" frame number : " << frames<< " time elapsed :" <<getMilliSpan(timeA)/1000<< " s " <<" number of collisions : " << possibleCollisions.size()<<endl;
			cout<<" integration number " << step << " of " << number_integrations <<endl;
			/*for (int i = 0; i < possibleCollisions.size(); i++)
			{
					int fiberIndexA = fiberIndices[((long)(possibleCollisions[i]>>32)) ];
					int segmentA = ((long)(possibleCollisions[i]>>32)) - segmentIndices[fiberIndexA];
					int fiberIndexB = fiberIndices[(long) possibleCollisions[i]];
					int segmentB = (long) possibleCollisions[i] - segmentIndices[fiberIndexB];

			cout<<" fiber A " << fiberIndexA << " segment " << segmentA <<" Fiber B " <<fiberIndexB;
				cout<<" segment " << segmentB<<endl;

			}
			*/


			timeA= getMilliCount();
			/*
			Hinge HingeA0 = fibers[0].hinges[0];
			Hinge HingeA1 = fibers[0].hinges[1];
			Hinge HingeB0 = fibers[1].hinges[0];
			Hinge HingeB1 = fibers[1].hinges[1];
			
			cout<<" segment A  : " << HingeA0.position.transpose() << " - " << HingeA1.position.transpose()<<endl;
		 cout<<" segment B  : " << HingeB0.position.transpose() << " - " << HingeB1.position.transpose()<<endl;
		cout<<" segment A Excluded volume force : "<< HingeA0.exluded_volume_force.transpose() << endl;
		cout<<" segment B Excluded volume force : "<< HingeB0.exluded_volume_force.transpose() << endl;
		cout<<" segment A Excluded volume torque : "<< HingeA0.excluded_volume_torque.transpose() << endl;
		cout<<" segment B Excluded volume torque : "<< HingeB0.excluded_volume_torque.transpose() << endl;
		
		cout<<endl;*/
		}
		
	}
	

	free(boundingBoxes);
	outputFile.close();

	std::cin.get();


  return 0;
    

}
