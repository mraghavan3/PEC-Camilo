#include "hip/hip_runtime.h"
#include "Excluded Volume.h"


struct Distance
{
	Vector3d Gab;
	Vector3d pb;
	double Gab_norm;
	Vector3d r;
	Vector3d r2;
	bool collisionCourse;

};




void Distance_point_point(Vector3d pointA, Vector3d pointB, Vector3d & g_ab, double & g_ab_norm){

	g_ab = pointA-pointB;
	g_ab_norm = sqrt(g_ab.dot(g_ab));
	
}

void Distance_segments(Vector3d hingeA0, Vector3d hingeA1, Vector3d hingeB0, Vector3d hingeB1, Vector3d & r, Vector3d & r2,Vector3d & g_ab, double & g_ab_norm, bool & collisionCourse  ){
	/*Vector3d pa = hingeA1 - hingeA0;
	Vector3d pb = hingeB1 - hingeB0; 
	

	double la = sqrt(pa.dot(pa));
	double lb = sqrt(pb.dot(pb));


	pa = pa/la;
	pb = pb/lb;
	double padotpa = pa.dot(pa);
	double pbdotpb = pb.dot(pb);
	double padotpb = pa.dot(pb);
	Vector3d vecDifference = hingeA0-hingeB0;

	
	double Sab = ( (vecDifference).dot(pb) * (padotpb- (vecDifference).dot(pa) ) ) / ( 1 - padotpb* padotpb ) ;
	double Sba = ( (-vecDifference).dot(pa) * (padotpb - (-vecDifference).dot(pb) ) ) / ( 1 - padotpb* padotpb ) ;
	

	/*
	double Sab = ( (hingeA0-hingeB0).dot(pb) * ( pa.dot(pb) - (hingeA0-hingeB0).dot(pa) ) ) / ( 1 - pa.dot(pb)* pa.dot(pb) ) ;
	double Sba = ( (hingeB0-hingeA0).dot(pa) * ( pb.dot(pa) - (hingeB0-hingeA0).dot(pa) ) ) / ( 1 - pb.dot(pa)* pb.dot(pa) ) ;
	
	r= pb * Sba;
	r2= pa * Sba;

	g_ab = hingeA0 + Sab*pa -hingeB0 - Sba*pb;
	g_ab_norm = sqrt(g_ab.dot(g_ab));
	collisionCourse =false;

	if (Sab <= la && Sab >= 0){
		if ( Sba <= lb && Sba >= 0){
			collisionCourse =true;
		}
	}
	*/
	// New test
	Vector3d u = hingeA1 - hingeA0;
	Vector3d v = hingeB1 - hingeB0;
	Vector3d w = hingeA0 - hingeB0;
	double a = u.dot(u);      // square of the length of segment A    
	double b = u.dot(v);
	double c = v.dot(v);      // square root of the length of segment B  
	double d = u.dot(w);
	double e = v.dot(w);
	double D = a*c - b*b;      

	double SMALL_NUM = 0.0000000001;
	double sc,tc;

    if (D < SMALL_NUM)  
	{
        sc = 0.0;
		if(b>c) {tc = d/b;} else {tc = e/c;} 
	}   
    else 
	{
        sc = (b*e - c*d) / D;
        tc = (a*e - b*d) / D;
	} 
    g_ab =( w + (sc * u) - (tc * v));
    g_ab_norm = sqrt(g_ab.dot(g_ab));

	r = -u * sc;
	r2 = -v * tc;
	if (sc <= 1 && sc >= 0){
		if ( tc <= 1 && tc >= 0){
			collisionCourse =true;
		}
	}
}

void Distance_point_segment(Vector3d rA , Vector3d rB, Vector3d  rB_end, Vector3d & r, Vector3d & g_ab, double & g_ab_norm, bool & collisionCourse ){

	Vector3d pb = rB_end - rB;
	double lb = sqrt(pb.dot(pb));
	pb = pb/lb;
	double Sba = (rA - rB).dot(pb);
	g_ab = rA -rB - Sba*pb;
	g_ab_norm = sqrt(g_ab.dot(g_ab));

	r= pb * Sba;
	collisionCourse =false;

	
		if ( Sba <= lb && Sba >= 0){
			collisionCourse =true;
		}
	

}




// compute excluded volume forces for debugging
void computeExludedVolumeForces( Hinge& HingeA0,  const Hinge& HingeA1, Hinge& HingeB0, const Hinge& HingeB1, double threshold, double Excl_vol_fac, double bead_radius, double & gab_min, vector<Distance> & Vec, bool isForNeighborsList, int timeStep){
	// TO DO, this is too verbose, make it more compact

	//vector<Distance> Vec(7);
	Distance tempVec;

	Distance_segments(HingeA0.position,HingeA1.position,HingeB0.position,HingeB1.position, tempVec.r, tempVec.r2, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	Vec[0]=tempVec; // 1

	Distance_point_segment(HingeA0.position, HingeB0.position, HingeB1.position, tempVec.r, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	tempVec.r2.setZero();
	Vec[1]=tempVec; // 2

	Distance_point_segment(HingeA1.position, HingeB0.position, HingeB1.position, tempVec.r, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	Vec[2]=tempVec; // 3

	Distance_point_point(HingeA0.position, HingeB0.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r.setZero();
	tempVec.r2.setZero();
	tempVec.collisionCourse =true; 
	Vec[3]=tempVec; // 4

	Distance_point_point(HingeA1.position, HingeB0.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r.setZero();
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	tempVec.collisionCourse =true;
	Vec[4]=tempVec; // 5

	Distance_point_point(HingeA0.position, HingeB1.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r = HingeB1.position-HingeB0.position;
	tempVec.r2.setZero();
	tempVec.collisionCourse =true;
	Vec[5]=tempVec; // 6

	Distance_point_point(HingeA1.position, HingeB1.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r = HingeB1.position-HingeB0.position;
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	tempVec.collisionCourse =true;
	Vec[6]=tempVec; // 7

	gab_min = Vec[6].Gab_norm;
	int k =6;

	for (int i = 0; i < 6; i++)
	{
		if (Vec[i].Gab_norm < gab_min  && Vec[i].collisionCourse)
		{
			gab_min = Vec[i].Gab_norm;
			k=i;

			


		}
	}
	
	if (!isForNeighborsList && gab_min < threshold && Vec[k].collisionCourse )
	{
		Vector3d Excluded_volume_partial_force = -Vec[k].Gab * Excl_vol_fac * exp(-2 *(Vec[k].Gab_norm / bead_radius -2));
		HingeB0.exluded_volume_force +=  Excluded_volume_partial_force ;
		HingeB0.excluded_volume_torque += Vec[k].r2.cross(Excluded_volume_partial_force);


		HingeA0.exluded_volume_force -=  Excluded_volume_partial_force ;
		HingeA0.excluded_volume_torque += Vec[k].r.cross(-Excluded_volume_partial_force);



		/*
		cout<<" segment A  : " << HingeA0.position.transpose() << " - " << HingeA1.position.transpose()<<endl;
		cout<<" segment B  : " << HingeB0.position.transpose() << " - " << HingeB1.position.transpose()<<endl;
		
		cout<<" segment A Excluded volume force : "<< HingeA0.exluded_volume_force.transpose() << endl;
		cout<<" segment B Excluded volume force : "<< HingeB0.exluded_volume_force.transpose() << endl;
		*/
	}

	if (timeStep% 1000 == 0)
	{
		cout<<" segment A  : " << HingeA0.position.transpose() << " - " << HingeA1.position.transpose()<<endl;
		cout<<" segment B  : " << HingeB0.position.transpose() << " - " << HingeB1.position.transpose()<<endl;
		cout<<"gab min " <<  gab_min<< "  Gab "<< Vec[k].Gab.transpose()<<endl;
		cout<<"r " <<  Vec[k].r.transpose() << "  r2 "<< Vec[k].r2.transpose()<<endl;
		switch (k)
			{
		case 0:
			cout<< " 0 segment-segment is shortest distance " <<endl;
			break;
		case 1:
			cout<< " 1 point-segment is shortest distance " <<endl;
			break;
		case 2:
			cout<< " 2 point-segment is shortest distance " <<endl;
			break;
		case 3:
			cout<< " 3 point-point is shortest distance " <<endl;
			break;
		case 4:
			cout<< " 4 point-point is shortest distance " <<endl;
			break;
		case 5:
			cout<< " 5 point-point is shortest distance " <<endl;
			break;
		case 6:
			cout<< " 6 point-point is shortest distance " <<endl;
		break;
			default:
				break;
			}
		cout<<endl;
	}
		

	


}






// TO DO Change type Vector3D to Type hinge // done
void computeExludedVolumeForces( Hinge& HingeA0,  const Hinge& HingeA1, Hinge& HingeB0, const Hinge& HingeB1, double threshold, double Excl_vol_fac, double bead_radius, double & gab_min, vector<Distance> & Vec, bool isForNeighborsList){  

	// TO DO, this is too verbose, make it more compact

	//vector<Distance> Vec(7);
	Distance tempVec;

	Distance_segments(HingeA0.position,HingeA1.position,HingeB0.position,HingeB1.position, tempVec.r, tempVec.r2, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	Vec[0]=tempVec; // 1

	Distance_point_segment(HingeA0.position, HingeB0.position, HingeB1.position, tempVec.r, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	tempVec.r2.setZero();
	Vec[1]=tempVec; // 2

	Distance_point_segment(HingeA1.position, HingeB0.position, HingeB1.position, tempVec.r, tempVec.Gab, tempVec.Gab_norm, tempVec.collisionCourse);
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	Vec[2]=tempVec; // 3

	Distance_point_point(HingeA0.position, HingeB0.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r.setZero();
	tempVec.r2.setZero();
	tempVec.collisionCourse =true; 
	Vec[3]=tempVec; // 4

	Distance_point_point(HingeA1.position, HingeB0.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r.setZero();
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	tempVec.collisionCourse =true;
	Vec[4]=tempVec; // 5

	Distance_point_point(HingeA0.position, HingeB1.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r = HingeB1.position-HingeB0.position;
	tempVec.r2.setZero();
	tempVec.collisionCourse =true;
	Vec[5]=tempVec; // 6

	Distance_point_point(HingeA1.position, HingeB1.position,tempVec.Gab, tempVec.Gab_norm);
	tempVec.r = HingeB1.position-HingeB0.position;
	tempVec.r2 =  HingeA1.position-HingeA0.position;
	tempVec.collisionCourse =true;
	Vec[6]=tempVec; // 7

	gab_min = Vec[6].Gab_norm;
	int k =6;

	for (int i = 0; i < 6; i++)
	{
		if (Vec[i].Gab_norm < gab_min  && Vec[i].collisionCourse)
		{
			gab_min = Vec[i].Gab_norm;
			k=i;

			


		}
	}
	/*
	if (isForNeighborsList)
	{
	switch (k)
			{
	case 0:
		cout<< " 0 segment-segment is shortest distance " <<endl;
		break;
	case 1:
		cout<< " 1 point-segment is shortest distance " <<endl;
		break;
	case 2:
		cout<< " 2 point-segment is shortest distance " <<endl;
		break;
	case 3:
		cout<< " 3 point-point is shortest distance " <<endl;
		break;
	case 4:
		cout<< " 4 point-point is shortest distance " <<endl;
		break;
	case 5:
		cout<< " 5 point-point is shortest distance " <<endl;
		break;
	case 6:
		cout<< " 6 point-point is shortest distance " <<endl;
		break;
			default:
				break;
			}
	}
	*/
	if (!isForNeighborsList && gab_min < threshold && Vec[k].collisionCourse )
	{
		Vector3d Excluded_volume_partial_force = -Vec[k].Gab * Excl_vol_fac * exp(-2 *(Vec[k].Gab_norm / bead_radius -2));
		HingeB0.exluded_volume_force +=  Excluded_volume_partial_force ;
		HingeA0.exluded_volume_force -=  Excluded_volume_partial_force ;
		HingeB0.excluded_volume_torque += Vec[k].r.cross(Excluded_volume_partial_force);
		HingeA0.excluded_volume_torque += Vec[k].r2.cross(-Excluded_volume_partial_force);
		/*
		cout<<" segment A  : " << HingeA0.position.transpose() << " - " << HingeA1.position.transpose()<<endl;
		cout<<" segment B  : " << HingeB0.position.transpose() << " - " << HingeB1.position.transpose()<<endl;
		cout<<" segment A Excluded volume force : "<< HingeA0.exluded_volume_force.transpose() << endl;
		cout<<" segment B Excluded volume force : "<< HingeB0.exluded_volume_force.transpose() << endl;
		*/
	}


}

double dotProduct(double Ax, double Ay, double Az, double Bx, double By, double Bz  ){

	return Ax*Bx + Ay*By + Az*Bz;

}

void findClosestPointSegmentDistance( double pointX, double pointY, double pointZ, double segmentAX, double segmentAY, double segmentAZ, double segmentBX, double segmentBY, double segmentBZ, double &distance, double &closestvectorX, double & closestvectorY){

	double ABX = segmentBX - segmentAX;
	double ABY = segmentBY - segmentAY;
	double ABZ = segmentBZ - segmentAZ;

	double t = dotProduct(  (pointX-segmentAX),(pointY-segmentAY),(pointZ-segmentAZ) , ABX , ABY, ABZ ) / dotProduct( ABX, ABY, ABZ , ABX, ABY, ABZ   );

	// clip it
	if (t<0){ t = 0;} 
	if (t>0){ t = 1;}

	double closestX = segmentAX + t * ABX;	  
	double closestY = segmentAY + t * ABY;
	double closestZ = segmentAZ + t * ABZ;

	closestvectorX = pointX-closestX;
	closestvectorY = pointY-closestY;


	 distance = sqrt( closestvectorX*closestvectorX + closestvectorY*closestvectorY    );
}


void calculateWallInteraction(Hinge & hinge1, double x1, double y1, double x2, double y2, double fiberRadius,  double excludedVolumeForce, double threshold){

	double closestDistance;
	double closestXVector;
	double closestYVector;


	findClosestPointSegmentDistance(hinge1.position[0],hinge1.position[1],0,x1,y1,0,x2,y2,0,closestDistance,closestXVector,closestYVector);
	//vector has correct sign, away from the wall
	if (closestDistance <= threshold)
	{
		

		hinge1.exluded_volume_force(0)+= closestXVector / closestDistance * excludedVolumeForce *  exp(-2 *(closestDistance / fiberRadius -2));
		hinge1.exluded_volume_force(1)+= closestYVector / closestDistance * excludedVolumeForce *  exp(-2 *(closestDistance / fiberRadius -2));
		//cout<<"Collision with wall, force in y is "<<hinge1.exluded_volume_force(1)<< "  closestDistance  "<<  closestDistance << " closestYVector "<<closestYVector << endl;
		//cout<< " x1  " <<x1<<" y1 " << y1<< " x2 "<<x2<<" y2 "<<y2<<" hinge x " <<hinge1.position[0]<< " hinge yx  " <<hinge1.position[1]<<endl;
	}


}



//collisions with walls
void interactionsWallsTri(vector<fiber> & fibers , int numNodes,vector<double> & meshPoints , vector<int> &meshConnectivity , vector<int> meshNeighbors , double fiberRadius , double excludedVolumeForce, double threshold, int nodesPerElement){
	#pragma omp parallel for
	for (int fiberID = 0; fiberID < fibers.size(); fiberID++)
	{
		for (int hingeID = 0; hingeID < fibers[fiberID].numberOfSegments; hingeID++)
		{
			for (int faceID = 0; faceID < 3; faceID++)// each face of the triangle
			{
				if (fibers[fiberID].hinges[hingeID].elementID!= -1 ) // the hinge is associated to an element
				{
					if (meshNeighbors[fibers[fiberID].hinges[hingeID].elementID*3+faceID ] ==-1){//-1 means that theres no neighbors in that face, thus is a boundary

						// the nodes that make that edge depndes on the index of the face:
						//face 0: nodes 0, 1, 
						//face 1: nodes 1, 2
						//face 2: nodes 2, 0
						int nodeA ;
							int nodeB;
						switch (faceID)
						{
						case 0:
							nodeA = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 0];
							nodeB = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 1];
							
							break;
						case 1:
							nodeA = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 1];
							nodeB = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 2];
								break;
							case 2:
								nodeA = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 2];
							nodeB = meshConnectivity[ fibers[fiberID].hinges[hingeID].elementID * nodesPerElement + 0];
								break;



						default:
							break;
						}

						calculateWallInteraction(fibers[fiberID].hinges[hingeID], meshPoints[ nodeA  ], meshPoints[ nodeA + numNodes  ],  meshPoints[ nodeB  ],meshPoints[ nodeB + numNodes  ],fiberRadius,excludedVolumeForce,threshold);
						

					}


				}
				



			}



		}
	}





}



void refineNeighbors(vector<fiber> & fibers, double beadRadius, thrust::host_vector<long long>  potentialCollisions, thrust::host_vector<int>  fiberIndices, thrust::host_vector<int>  segmentIndices) {

	
	double threshold = 4  * beadRadius;
	double Excl_volume_face =1000000;
	vector<Distance> Vec(7); // used in compute exluded forces
	//cout<< " the threshold is " << threshold <<endl;
	// for every fiber
	int numberOFCollisionsMy= 0;
	int timeA;
	
	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
				 timeA =getMilliCount();
	#endif
	/*
	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		// for every segment of fiber i
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			
			// lets uppdate the neighbor list, naive programing, will be improved afterwards 
			// I dont know exactly how it will end up looking like
			vector<int> newNeighborFiberIndex;
			vector<int> newNeighborSegmentIndex;
			int newNumberOfNeighbours=0;
			for (unsigned int k = 0; k < fibers[i].segments[j].neighboringFibers.size(); k++)
			{

				newNeighborFiberIndex.resize(fibers[i].segments[j].neighboringFibers.size());
				newNeighborSegmentIndex.resize(fibers[i].segments[j].neighboringFibers.size());

				double min_Gab;
				int neighbouringFiberIndex =fibers[i].segments[j].neighboringFibers[k];
				int neighbouringSegmentIndex = fibers[i].segments[j].neighboringSegment[k];
				//TODO add flag for neighbors list
				computeExludedVolumeForces(fibers[i].hinges[j], fibers[i].hinges[j+1], fibers[neighbouringFiberIndex].hinges[neighbouringSegmentIndex], 
					fibers[neighbouringFiberIndex].hinges[neighbouringSegmentIndex+1], threshold, Excl_volume_face ,beadRadius, min_Gab, Vec,true);
				if (neighbouringFiberIndex != i && min_Gab < threshold  ) // != for non unique interactions, > for unique interactions
				{
					newNeighborFiberIndex[newNumberOfNeighbours] = neighbouringFiberIndex;
					newNeighborSegmentIndex[newNumberOfNeighbours] = neighbouringSegmentIndex;
					//newNeighborFiberIndex.push_back(neighbouringFiberIndex);
					//newNeighborSegmentIndex.push_back(neighbouringSegmentIndex);
					newNumberOfNeighbours++;
					numberOFCollisionsMy ++;

				}else
				{
					//cout<< " neighbor dismissed " <<endl;
				}
			}
			newNeighborFiberIndex.resize(newNumberOfNeighbours);
			newNeighborSegmentIndex.resize(newNumberOfNeighbours);

			//cout<< "1.  number of neighbours "<< fibers[i].segments[j].neighboringFibers.size() << " new number " << newNeighborFiberIndex.size()<<endl;
			fibers[i].segments[j].neighboringFibers.clear();
			fibers[i].segments[j].neighboringFibers = newNeighborFiberIndex;
			fibers[i].segments[j].neighboringSegment.clear();
			fibers[i].segments[j].neighboringSegment = newNeighborSegmentIndex;
			//cout<< "1.  number of neighbours "<< fibers[i].segments[j].neighboringFibers.size() << " new number " << newNeighborFiberIndex.size()<<endl;
		}
	}
	

	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
	cout << "time required to find neighbors my method " << getMilliSpan(timeA)<<" ms"<<endl;
	#endif
	*/

	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
				 timeA =getMilliCount();
	#endif

	thrust::host_vector<int> stencil;

	for (unsigned int i = 0; i < potentialCollisions.size(); i++)
	{
		int particleA = (long)(potentialCollisions[i]>>32); // global index of particle A
		int particleB = (long) potentialCollisions[i]; // global index of particle B
		int particleAfiberIndex = fiberIndices[particleA]; // fiber to which particle A belongs
		int particleBfiberIndex = fiberIndices[particleB]; // fiber to which particle B belongs
		int particleASegmentIndex = particleA - segmentIndices[particleAfiberIndex];// local index of particle A
		int particleBSegmentIndex = particleB - segmentIndices[particleBfiberIndex];// local index of particle B

		double min_Gab;
		computeExludedVolumeForces(fibers[particleAfiberIndex].hinges[particleASegmentIndex], fibers[particleAfiberIndex].hinges[particleASegmentIndex+1], 
						fibers[particleBfiberIndex].hinges[particleBSegmentIndex], fibers[particleBfiberIndex].hinges[particleBSegmentIndex+1],
						threshold, Excl_volume_face ,beadRadius, min_Gab, Vec,true);

		
		//cout<<"min_gab " << min_Gab << "threshold " <<threshold<<endl;
		if (min_Gab < threshold  )
				{
					
					stencil.push_back(1);
					//cout<<"do not delete interaction " <<endl;
				}
		else {
			stencil.push_back(0);
			//cout<<" delete interaction " <<endl;
		}

		 

	}
	Broadphase broadphaseManager;

	
	
	if (potentialCollisions.size() != 0 )
	{
		int numberOfCollisions = broadphaseManager.removeSameFiberCollisionsH(potentialCollisions,stencil);
		int realCollissions = stencil.size() - numberOfCollisions;
		potentialCollisions.resize(realCollissions);
	}
	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
	cout << "time required to find neighbors broadphase method " << getMilliSpan(timeA)<<" ms"<<endl;
	#endif

	
}




void calculateFiberInteractions(vector<fiber> & fibers, double beadRadius) {

	
	double threshold = 10 * beadRadius;
	double Excl_volume_face =100;
	vector<Distance> Vec(7); // used in compute exluded forces
	//cout<< " the threshold is " << threshold <<endl;
	// for every fiber

	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		for (unsigned int j = 0; j < fibers[i].numberOfHinges; j++)
		{
			fibers[i].hinges[j].exluded_volume_force.fill(0);
			fibers[i].hinges[j].torque.fill(0);
			fibers[i].hinges[j].excluded_volume_torque.fill(0);
		}
	}


	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		// for every segment of fiber i
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			
			for (unsigned int k = 0; k < fibers[i].segments[j].neighboringFibers.size(); k++)
			{


				double min_Gab;
				int neighbouringFiberIndex =fibers[i].segments[j].neighboringFibers[k];
				int neighbouringSegmentIndex = fibers[i].segments[j].neighboringSegment[k];
				if (neighbouringFiberIndex != i) // If the neighbor does not belong to the same fiber as the current one
				{
				//TODO add flag for neighbors list // done
				computeExludedVolumeForces(fibers[i].hinges[j], fibers[i].hinges[j+1], fibers[neighbouringFiberIndex].hinges[neighbouringSegmentIndex], 
					fibers[neighbouringFiberIndex].hinges[neighbouringSegmentIndex+1], threshold, Excl_volume_face ,beadRadius, min_Gab,Vec,false);
				}
				
	
			}
			
		}
	}
}


void calculateFiberInteractions(vector<fiber> & fibers, double beadRadius, thrust::host_vector<long long>  potentialCollisions, thrust::host_vector<int>  fiberIndices, thrust::host_vector<int>  segmentIndices  ){

	double threshold = 2.1 * beadRadius;
	double Excl_volume_face =100;
	vector<Distance> Vec(7); // used in compute exluded forces
	//cout<< " the threshold is " << threshold <<endl;
	// for every fiber

	#ifdef PRINT_DEBUG_INFO
	cout<<" make sure that everything is zeroed "<<endl;
	#endif

	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfHinges; j++)
		{
			fibers[i].hinges[j].exluded_volume_force.fill(0);
			fibers[i].hinges[j].torque.fill(0);
			fibers[i].hinges[j].excluded_volume_torque.fill(0);

			#ifdef PRINT_DEBUG_INFO
			cout<<"fiber " << i << " segment " << j << endl;
			cout<< " excluded volume force " << fibers[i].hinges[j].exluded_volume_force<<endl;
			cout<< " torque " << fibers[i].hinges[j].torque<<endl;
			cout<< " excluded volume torque " << fibers[i].hinges[j].excluded_volume_torque<<endl;
			#endif

		}
	}


	for (unsigned int i = 0; i < potentialCollisions.size(); i++)
	{
		int particleA = (long)(potentialCollisions[i]>>32); // global index of particle A
		int particleB = (long) potentialCollisions[i]; // global index of particle B
		int particleAfiberIndex = fiberIndices[particleA]; // fiber to which particle A belongs
		int particleBfiberIndex = fiberIndices[particleB]; // fiber to which particle B belongs
		int particleASegmentIndex = particleA - segmentIndices[particleAfiberIndex];// local index of particle A
		int particleBSegmentIndex = particleB - segmentIndices[particleBfiberIndex];// local index of particle B
		bool isForNeighborsList = false;
		double min_Gab;
		computeExludedVolumeForces(fibers[particleAfiberIndex].hinges[particleASegmentIndex], fibers[particleAfiberIndex].hinges[particleASegmentIndex+1], 
						fibers[particleBfiberIndex].hinges[particleBSegmentIndex], fibers[particleBfiberIndex].hinges[particleBSegmentIndex+1],
						threshold, Excl_volume_face ,beadRadius, min_Gab, Vec,isForNeighborsList);

		#ifdef PRINT_DEBUG_INFO
		cout<< "fiber " <<particleAfiberIndex <<" segment "<<particleASegmentIndex<<" force "<<	fibers[particleAfiberIndex].hinges[particleASegmentIndex].exluded_volume_force <<endl;
		cout<< "fiber " <<particleBfiberIndex <<" segment "<<particleBSegmentIndex<<" force "<<	fibers[particleBfiberIndex].hinges[particleBSegmentIndex].exluded_volume_force <<endl;

		#endif



	}



	#ifdef PRINT_DEBUG_INFO
	cout<<" results form calculations"<<endl;
	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfHinges; j++)
		{
			cout<<"fiber " << i << " segment " << j << endl;
			cout<< " excluded volume force " << fibers[i].hinges[j].exluded_volume_force<<endl;
			cout<< " torque " << fibers[i].hinges[j].torque<<endl;
			cout<< " excluded volume torque " << fibers[i].hinges[j].excluded_volume_torque<<endl;
		}
	}
	#endif

}


void calculateFiberInteractions(vector<fiber> & fibers, double beadRadius, thrust::host_vector<long long>  potentialCollisions, thrust::host_vector<int>  fiberIndices, thrust::host_vector<int>  segmentIndices , int timeStep ){

	double threshold = 2.1 * beadRadius;

	

	double Excl_volume_face =100;
	vector<Distance> Vec(7); // used in compute exluded forces
	//cout<< " the threshold is " << threshold <<endl;
	// for every fiber

	#ifdef PRINT_DEBUG_INFO
	cout<<" make sure that everything is zeroed "<<endl;
	#endif

	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfHinges; j++)
		{
			fibers[i].hinges[j].exluded_volume_force.fill(0);
			fibers[i].hinges[j].torque.fill(0);
			fibers[i].hinges[j].excluded_volume_torque.fill(0);

			#ifdef PRINT_DEBUG_INFO
			cout<<"fiber " << i << " segment " << j << endl;
			cout<< " excluded volume force " << fibers[i].hinges[j].exluded_volume_force<<endl;
			cout<< " torque " << fibers[i].hinges[j].torque<<endl;
			cout<< " excluded volume torque " << fibers[i].hinges[j].excluded_volume_torque<<endl;
			#endif

		}
	}


	for (unsigned int i = 0; i < potentialCollisions.size(); i++)
	{
		int particleA = (long)(potentialCollisions[i]>>32); // global index of particle A
		int particleB = (long) potentialCollisions[i]; // global index of particle B
		int particleAfiberIndex = fiberIndices[particleA]; // fiber to which particle A belongs
		int particleBfiberIndex = fiberIndices[particleB]; // fiber to which particle B belongs
		int particleASegmentIndex = particleA - segmentIndices[particleAfiberIndex];// local index of particle A
		int particleBSegmentIndex = particleB - segmentIndices[particleBfiberIndex];// local index of particle B
		bool isForNeighborsList = false;
		double min_Gab;
		computeExludedVolumeForces(fibers[particleAfiberIndex].hinges[particleASegmentIndex], fibers[particleAfiberIndex].hinges[particleASegmentIndex+1], 
						fibers[particleBfiberIndex].hinges[particleBSegmentIndex], fibers[particleBfiberIndex].hinges[particleBSegmentIndex+1],
						threshold, Excl_volume_face ,beadRadius, min_Gab, Vec,isForNeighborsList,timeStep);

		#ifdef PRINT_DEBUG_INFO
		cout<< "fiber " <<particleAfiberIndex <<" segment "<<particleASegmentIndex<<" force "<<	fibers[particleAfiberIndex].hinges[particleASegmentIndex].exluded_volume_force <<endl;
		cout<< "fiber " <<particleBfiberIndex <<" segment "<<particleBSegmentIndex<<" force "<<	fibers[particleBfiberIndex].hinges[particleBSegmentIndex].exluded_volume_force <<endl;

		#endif



	}



	#ifdef PRINT_DEBUG_INFO
	cout<<" results form calculations"<<endl;
	for (unsigned int i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfHinges; j++)
		{
			cout<<"fiber " << i << " segment " << j << endl;
			cout<< " excluded volume force " << fibers[i].hinges[j].exluded_volume_force<<endl;
			cout<< " torque " << fibers[i].hinges[j].torque<<endl;
			cout<< " excluded volume torque " << fibers[i].hinges[j].excluded_volume_torque<<endl;
		}
	}
	#endif

}