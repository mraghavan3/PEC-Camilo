#include "hip/hip_runtime.h"
#include "Find Neighbors.cuh"
#include "Broadphase.h"

void getMaxMin( double a, double b , double &max , double &min ){

	if (a > b )
			{
				max = a;
				min = b;
			}
			else
			{
				min = a;
				max = b;
			}

}

void appendVector( vector<int> & destinationVector, vector<int> sourceVector){
	int initialSize = destinationVector.size();
	destinationVector.resize(initialSize + sourceVector.size()); // added to improve efficiency
	for (uint i = 0; i < sourceVector.size(); i++)
	{
		destinationVector[initialSize + i ] = sourceVector[i];
		//destinationVector.push_back(sourceVector[i]); //removed to improve efficiency
	}
}


void findNeighbours( vector<fiber> &fibers, double fiberRadius, thrust::host_vector<long long> & potentialCollisions, thrust::host_vector<int> fiberIndices){
	vector<Cell> cells ;
	
	
	int timeA = getMilliCount();

	// first step, find box dimensions, and cell dimension rc
	// I don't like having to do this, it seems like alot of operations to find rc
	
	double maxSegmentLength=0;
	double minX  = fibers[0].hinges[0].position(0);
	double minY = fibers[0].hinges[0].position(1);
	double minZ = fibers[0].hinges[0].position(2);
	double maxX = fibers[0].hinges[0].position(0);
	double maxY = fibers[0].hinges[0].position(1);
	double maxZ = fibers[0].hinges[0].position(2);
	
	for (uint i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j <fibers[i].numberOfSegments ; j++)
		{
			//compute length of each segment
			
			double segmentLength = sqrt(  (fibers[i].hinges[j].position-fibers[i].hinges[j+1].position).dot(fibers[i].hinges[j].position-fibers[i].hinges[j+1].position));
			
			// record the longest segment
			if (segmentLength>maxSegmentLength){maxSegmentLength = segmentLength;}
			
			// compute midpoint of current segment
			
			fibers[i].midpoints[j] = (fibers[i].hinges[j].position + fibers[i].hinges[j+1].position)/2;
			
			//record the minimum and maximum coords of the fibers
			if (minX > fibers[i].midpoints[j](0)){minX = fibers[i].midpoints[j](0);}
			if (minY > fibers[i].midpoints[j](1)){minY = fibers[i].midpoints[j](1);}
			if (minZ > fibers[i].midpoints[j](2)){minZ = fibers[i].midpoints[j](2);}
			if (maxX < fibers[i].midpoints[j](0)){maxX = fibers[i].midpoints[j](0);}
			if (maxY < fibers[i].midpoints[j](1)){maxY = fibers[i].midpoints[j](1);}
			if (maxZ < fibers[i].midpoints[j](2)){maxZ = fibers[i].midpoints[j](2);}

		}
	}
	
	// this is the characteristic length of the cell 
	double rc =maxSegmentLength;
	
	// these are the dimensions of the box
	Vector3d boxLength;//L
	boxLength<< (maxX-minX), (maxY - minY) , (maxZ-minZ) ;
	
	//cout<<endl<< "box dimensions" << boxLength;
	//cout<<endl<< "characteristic dimension " << rc;
	//cout<<endl<< " minx " << minX << " maxx " << maxX << endl;
	
	// the number of cells in each direction
	Vector3i cellsNumber; // Lc
	cellsNumber<< (int)(floor(boxLength(0)/rc)) , (int)(floor(boxLength(1)/rc)) , (int)(floor(boxLength(2)/rc)) ;
	for (int i = 0; i < 3; i++)
	{
		if (cellsNumber(i) ==0)
		{
			cellsNumber(i) =1;
		}
	}


	// 
	Vector3d cellLength; // rc
	cellLength<< (boxLength(0)/cellsNumber(0)) , (boxLength(1)/cellsNumber(1)) , (boxLength(2)/cellsNumber(2)) ;
	/*

	// 
	int totalNumberCells = cellsNumber(0)*cellsNumber(1)*cellsNumber(2);
	//cout<<" totalNumber of cells" << totalNumberCells<<endl;
	cells.resize(totalNumberCells); // For a reason I cant understand I have to have this line and the for that follows, 
									// otherwise I'll get an error that I do not know what does it mean;
	//cout<< endl << "number of cells " << cellsNumber(0)<< " y " <<cellsNumber(1) << " z " <<cellsNumber(2)<<endl;
	
	//cout<<"Number of cells " << cells.size() << " Number of fibers " << fibers.size()<< endl;


	for (int i = 0; i < totalNumberCells+1; i++)
	{
		Cell tempcell;
		tempcell.hasFibers =false;
		//cells[i] = tempcell;
		tempcell.fiberIndex.clear();

		cells.push_back(tempcell);
		cells[i].hasFibers=false;
		
	}

	 // Lets find the cell where each segment is 

	for (uint i = 0; i < fibers.size(); i++)
	{
		//cout<< "fiber " << i << " of " << fibers.size() << endl;
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			
			Vector3i cellID;
			cellID(0) = floor( (fibers[i].midpoints[j].x() - minX) / cellLength.x()); 
			cellID(1) = floor( (fibers[i].midpoints[j].y() - minY) / cellLength.y()); 
			cellID(2) = floor( (fibers[i].midpoints[j].z() - minZ) / cellLength.z()); 

			//cout<< " index of segment x " << cellID.x() << " y " <<cellID.y() << " z "<< cellID.z() << endl;
			int cellIndex = cellID.x() * cellsNumber.y() * cellsNumber.z() + cellID.y() * cellsNumber.z() + cellID.z();
			//cout<< "segment is on cell number : " << cellIndex << " max index "<<  totalNumberCells  << endl; 
			cells[cellIndex].hasFibers =true;
			//cout<< " done with this cycle "<<endl;
			cells[cellIndex].fiberIndex.push_back(i); // I need to pushback because  I dont know the size
			//cells[cellIndex].a =2;
			cells[cellIndex].segmentIndex.push_back(j);
			fibers[i].segments[j].cellID = cellIndex;


			//
		}

	}
	

	// Lets find the neighbors of every cell
	// iterating over the three dimensional indices of the cells
	for (int xIndex = 0; xIndex < cellsNumber.x(); xIndex++)
	{
		for (int yIndex = 0; yIndex < cellsNumber.y(); yIndex++)
		{
			for (int zIndex = 0; zIndex < cellsNumber.z(); zIndex++)
			{
				// name of variable says it all
				int currentCellIndex = xIndex * cellsNumber.y() * cellsNumber.z() + yIndex * cellsNumber.z() + zIndex;
				//cout<< " current index " << currentCellIndex<<endl;
				int numberOFneighboringCells =0 ;
				if ( cells[currentCellIndex].hasFibers){
					

				// Now I have to look for the 26 neighboring cells

					for (int xNeighbor = xIndex-1; xNeighbor < xIndex+2; xNeighbor++)
					{
						if (xNeighbor >= 0 && xNeighbor < cellsNumber.x()) // the index needs to be bigger than or equal to zero
																		   // And smaller than the number of cell in x direction
						{
							for (int yNeighbor = yIndex-1; yNeighbor < yIndex+2; yNeighbor++)
							{
								if (yNeighbor >= 0 && yNeighbor < cellsNumber.y())
								{
									for (int zNeighbor = zIndex-1; zNeighbor < zIndex + 2; zNeighbor++)
									{
										if (zNeighbor >= 0 && zNeighbor < cellsNumber.z())
										{
											int neighborCellIndex = xNeighbor * cellsNumber.y() * cellsNumber.z() + yNeighbor * cellsNumber.z() + zNeighbor;
											if (cells[neighborCellIndex].hasFibers)
											{
												//copy fiber index vector and segment index vector of the neighbor cell to the neighbor fiber array
												//and neighbor index vector
												appendVector(cells[currentCellIndex].neighborFiberIndex , cells[neighborCellIndex].fiberIndex);
												appendVector(cells[currentCellIndex].neighborSegmentIndex , cells[neighborCellIndex].segmentIndex);
											}
											
											if (neighborCellIndex!= currentCellIndex)
											{
												numberOFneighboringCells++;
											}
										}
									}
								}
							}
						}
					}
				}//ends first if
				//cout<< " number of neighboring cells " << numberOFneighboringCells << endl;
			}
		}
	}

	//cout<< " total number of cells " << totalNumberCells; 

	
 // copy the list of neighbors to each segment

	for (uint i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
		 fibers[i].segments[j].neighboringFibers.resize(0);
		 fibers[i].segments[j].neighboringSegment.resize(0);
		 appendVector(fibers[i].segments[j].neighboringFibers, cells[fibers[i].segments[j].cellID].neighborFiberIndex);
		 appendVector(fibers[i].segments[j].neighboringSegment, cells[fibers[i].segments[j].cellID].neighborSegmentIndex);
		}
	}

	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
	cout << "time required to find neighbors my method " << getMilliSpan(timeA)<<" ms"<<endl;
	#endif


	*/

	// Broad phase use
	
	
	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
				 timeA =getMilliCount();
	#endif

	// Step 1, get AABB data

	thrust::host_vector<real3> aabb_data_H;
	//custom_vector<real3> aabb_data;
	potentialCollisions.clear();

	double offsetFactor=1.5;


	// first half of the array has the minimum point of the bounding box, that part will be filled with the followin nested loops
	for (uint i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			double minCoord;
			double maxCoord;
			real3 temp;

			getMaxMin(fibers[i].hinges[j].position.x(),fibers[i].hinges[j+1].position.x() ,maxCoord,minCoord);
			temp.x = minCoord - offsetFactor*fiberRadius;
			getMaxMin(fibers[i].hinges[j].position.y(),fibers[i].hinges[j+1].position.y() ,maxCoord,minCoord);
			temp.y = minCoord - offsetFactor*fiberRadius;
			getMaxMin(fibers[i].hinges[j].position.z(),fibers[i].hinges[j+1].position.z() ,maxCoord,minCoord);
			temp.z = minCoord - offsetFactor*fiberRadius;
			aabb_data_H.push_back(temp);
			

		}
	}
	for (uint i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			double minCoord;
			double maxCoord;
			real3 temp;

			getMaxMin(fibers[i].hinges[j].position.x(),fibers[i].hinges[j+1].position.x() ,maxCoord,minCoord);
			temp.x = maxCoord + offsetFactor*fiberRadius;
			getMaxMin(fibers[i].hinges[j].position.y(),fibers[i].hinges[j+1].position.y() ,maxCoord,minCoord);
			temp.y = maxCoord + offsetFactor*fiberRadius;
			getMaxMin(fibers[i].hinges[j].position.z(),fibers[i].hinges[j+1].position.z() ,maxCoord,minCoord);
			temp.z = maxCoord + offsetFactor*fiberRadius;
			aabb_data_H.push_back(temp);

		}
	}

	#ifdef SIM_ENABLE_GPU_MODE
		custom_vector<real3> aabb_data = aabb_data_H;
	#else
		
	#endif
	
	
	//Step 2 run broadphase algorithm
	
	Broadphase broadphaseManager;
	broadphaseManager.setBinsPerAxis(make_real3(cellsNumber.x(),cellsNumber.y(),cellsNumber.z()));

	#ifdef SIM_ENABLE_GPU_MODE
		custom_vector<long long> potentialCollisions_D  = potentialCollisions;
		broadphaseManager.detectPossibleCollisions(aabb_data, potentialCollisions_D);
		potentialCollisions = potentialCollisions_D;
	#else
		broadphaseManager.detectPossibleCollisions(aabb_data_H, potentialCollisions);
	#endif
	


	/*cout<<"Number of possible contacts"<< broadphaseManager.getNumPossibleContacts()<<endl;
	cout<< " number of neighboring fibers" << fibers[0].segments[0].neighboringFibers.size()<< endl;
	cout<< " number of neighboring fibers" << fibers[0].segments[0].neighboringSegment.size()<< endl;
	//cout << "time required to find neighbors with broadphase 1 " << getMilliSpan(timeA)<<" ms"<<endl;
	//Remove collisions beteen segments of the same fiber*/

	thrust::host_vector<int> stencil_H;
	
	
	for (uint i = 0; i < broadphaseManager.getNumPossibleContacts(); i++)
	{
		long particleA = ((long)(potentialCollisions[i]>>32));
		long particleB = (long) potentialCollisions[i];
		if (fiberIndices[particleA] == fiberIndices[particleB])
		{
			stencil_H.push_back(1);
			//cout<<" Same fiber neighbors " <<endl;
		}
		else{
			stencil_H.push_back(0);
		}

	}

	custom_vector<int> stencil= stencil_H;

	#ifdef PRINT_DEBUG_INFO
	cout<<"number of possible collisions " << potentialCollisions.size()<<endl;
	cout << "time required to find neighbors with broadphase 1 " << getMilliSpan(timeA)<<" ms"<<endl;
	#endif
/*	for (int i = 0; i < broadphaseManager.getNumPossibleContacts(); i++)
	{


		//cout<<"My Method "<<" fiber "<<fibers[0].segments[2].neighboringFibers[i] <<" Segment "<< fibers[0].segments[2].neighboringSegment[i]<<endl;

		int fiberIndexA = fiberIndices[((long)(potentialCollisions[i]>>32)) ];
		int segmentA = ((long)(potentialCollisions[i]>>32)) - segmentIndex[fiberIndexA];
		int fiberIndexB = fiberIndices[(long) potentialCollisions[i]];
		int segmentB = (long) potentialCollisions[i] - segmentIndex[fiberIndexB];

		cout<<"BroadPhase,  element A  " <<  ((long)(potentialCollisions[i]>>32)) << " element B " << (long) potentialCollisions[i]<< " stencil: " << stencil[i]<<endl;
		cout<<" fiber A " << fiberIndexA << " segment " << segmentA <<" Fiber B " <<fiberIndexB;
				cout<<" segment " << segmentB<<endl;
	}*/


	int numberOfCollisions;

	#ifdef SIM_ENABLE_GPU_MODE
	//custom_vector<long long> potentialCollisions_D  = potentialCollisions;
	if (potentialCollisions_D.size() != 0)
	{
		numberOfCollisions = broadphaseManager.removeSameFiberCollisions(potentialCollisions_D,stencil);
		potentialCollisions = potentialCollisions_D;

		int realCollissions = stencil_H.size() - numberOfCollisions;
		potentialCollisions.resize(realCollissions);

		/*
		cout<<"Stencil : ";
		for (int i = 0; i < stencil_H.size(); i++)
		{
			cout<<stencil_H[i] << ", ";
		}
		cout<<endl;*/
	}
		
		
	#else
	if (potentialCollisions.size() != 0 )
	{
		numberOfCollisions = broadphaseManager.removeSameFiberCollisionsH(potentialCollisions,stencil);
	}
		
	#endif

	
	
	#ifdef PRINT_DEBUG_INFO_NEIGHBORS_TIMING
	 cout << "time required to find neighbors with broadphase" << getMilliSpan(timeA)<<" ms"<<endl;
	#endif



}
       





void findBoundingBoxes( vector<fiber> fibers, double * &boundingBox, double fiberRadius){

	// bounding box is an array with the format:
	// minx miny minz maxx maxy maxz

	double minx, miny , minz, maxx, maxy, maxz;
	int index =0;

	// loops over every hinge of the system
	for (uint i = 0; i < fibers.size(); i++)
	{
		for (int j = 0; j < fibers[i].numberOfSegments; j++)
		{
			
			// compares each pair of coordinate components to find out whis is the minimum and maximum
			// 
			getMaxMin(fibers[i].hinges[j].position.x() , fibers[i].hinges[j+1].position.x(), maxx , minx); 
			getMaxMin(fibers[i].hinges[j].position.y() , fibers[i].hinges[j+1].position.y(), maxy , miny);
			getMaxMin(fibers[i].hinges[j].position.z() , fibers[i].hinges[j+1].position.z(), maxx , minz);


			// stores the data
			boundingBox[index] = minx - fiberRadius; index++;
			boundingBox[index] = miny - fiberRadius; index++;
			boundingBox[index] = minz - fiberRadius; index++;
			boundingBox[index] = maxx + fiberRadius; index++;
			boundingBox[index] = maxy + fiberRadius; index++;
			boundingBox[index] = maxz + fiberRadius; index++;
		}
	}


}
